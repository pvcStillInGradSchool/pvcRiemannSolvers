#include "hip/hip_runtime.h"
// Copyright 2024 PEI Weicheng

#include <algorithm>
#include <concepts>
#include <cstdio>
#include <fstream>
#include <numeric>
#include <unordered_map>
#include <vector>

#include "mini/algebra/eigen.hpp"
#include "mini/mesh/vtk.hpp"

#include <CGAL/Simple_cartesian.h>
#include <CGAL/point_generators_3.h>
#include <CGAL/Projection_traits_xy_3.h>
#include <CGAL/Delaunay_triangulation_2.h>

#include <hip/hip_runtime_api.h>
#include <>

#include "yaml-cpp/yaml.h"

#define NSTREAMS 4

template <typename Delaunay>
std::vector<std::array<int, 3>> GetFaces(Delaunay const &delaunay) {
  // prepare the vertex_handle-to-vertex_index map:
  auto vertex_map = std::unordered_map<
      typename Delaunay::Vertex_handle, int>();
  int i = 0;  // index of vertex
  for (auto vertex_handle : delaunay.finite_vertex_handles()) {
    vertex_map[vertex_handle] = i++;
  }
  assert(delaunay.number_of_vertices() == i);
  // build faces (triangles)
  std::vector<std::array<int, 3>> faces;
  faces.reserve(delaunay.number_of_faces());
  for (auto face_handle : delaunay.finite_face_handles()) {
    auto &face = faces.emplace_back();
    face[0] = vertex_map.at(face_handle->vertex(0));
    face[1] = vertex_map.at(face_handle->vertex(1));
    face[2] = vertex_map.at(face_handle->vertex(2));
  }
  assert(faces.size() == delaunay.number_of_faces());
  return faces;
}

std::vector<std::array<int, 2>> GetEdges(std::vector<std::array<int, 3>> const &faces) {
  auto edges = std::vector<std::array<int, 2>>();
  for (auto [a, b, c] : faces) {
    auto emplace_back = [&edges](int i, int j) {
      auto &edge = edges.emplace_back();
      edge[0] = std::min(i, j);
      edge[1] = std::max(i, j);
    };
    emplace_back(a, b); emplace_back(b, c); emplace_back(c, a);
  }
  std::ranges::sort(edges);
  auto ret = std::ranges::unique(edges);
  edges.erase(ret.begin(), ret.end());
  return edges;
}

/**
 * @brief The distance function of a rectangle.
 * 
 */
template <std::floating_point Real>
Real Rectangle(Real x, Real y, Real x_min, Real x_max, Real y_min, Real y_max) {
  return -std::min(std::min(y - y_min, y_max - y),
                   std::min(x - x_min, x_max - x));
}

/**
 * @brief The distance function of a circle.
 * 
 */
template <std::floating_point Real>
__device__ __host__
Real Circle(Real x, Real y, Real x_center, Real y_center, Real radius) {
  return std::hypot(x - x_center, y - y_center) - radius;
}

/**
 * @brief The distance function of \f$ A \setminus B \f$.
 * 
 */
template <std::floating_point Real>
auto Difference(Real a, Real b) {
  return std::max(a, -b);
}

int n_fixed = 4;
double x_center = 0.0, y_center = 0.0, radius = 0.5;
double x_min = x_center - 1.0;
double x_max = -x_min;
double y_min = y_center - 1.0;
double y_max = -y_min;

template <std::floating_point Real>
Real distance(Real a, Real b) {
  return Difference(
      Rectangle(a, b, x_min, x_max, y_min, y_max),
      Circle(a, b, x_center, y_center, radius));
}

template <std::floating_point Real>
__device__ __host__
Real scaling(Real a, Real b) {
  return 0.05 + 0.3 * Circle(a, b, 0.0, 0.0, 0.5);
};

template <class Vector, class Distance>
int RejectPoints(Vector *x, Vector *y, Distance &&distance) {
  int n = x->size();
  assert(n == y->size());
  auto valid = std::vector<int>();
  for (int i = 0; i < n; i++) {
    if (distance((*x)[i], (*y)[i]) <= 0) {
      valid.push_back(i);
    }
  }
  int n_new = valid.size();
  Vector x_new(n_new), y_new(n_new);
  for (int i_new = 0; i_new < n_new; i_new++) {
    int i = valid[i_new];
    x_new[i_new] = (*x)[i];
    y_new[i_new] = (*y)[i];
  }
  std::swap(x_new, *x);
  std::swap(y_new, *y);
  return n_new;
}

template <class Vector, class Distance>
int RejectFaces(std::vector<std::array<int, 3>> *faces,
    Vector const &x, Vector const &y, Distance &&distance,
    decltype(x[0]) eps) {
  int n = faces->size();
  auto valid = std::vector<int>();
  for (int i = 0; i < n; i++) {
    auto [a, b, c] = faces->at(i);
    auto x_center = (x[a] + x[b] + x[c]) / 3.;
    auto y_center = (y[a] + y[b] + y[c]) / 3.;
    if (distance(x_center, y_center) + eps <= 0.) {
      valid.push_back(i);
    }
  }
  int n_new = valid.size();
  std::vector<std::array<int, 3>> faces_new(n_new);
  for (int i_new = 0; i_new < n_new; i_new++) {
    faces_new[i_new] = faces->at(valid[i_new]);
  }
  std::swap(faces_new, *faces);
  return n_new;
}

template <std::floating_point Real, int kNodes, class Distance>
void WriteVtu(std::string const &filename, bool binary,
    int n_point, Real const *x, Real const *y,  Real const *z,
    std::vector<std::array<int, kNodes>> const &cells,
    mini::mesh::vtk::CellType vtk_cell_type,
    Distance &&distance) {
  std::string endianness
      = (std::endian::native == std::endian::little)
      ? "\"LittleEndian\"" : "\"BigEndian\"";
  auto format = binary ? "\"binary\"" : "\"ascii\"";
  int n_cell = cells.size();
  assert(kNodes == mini::mesh::vtk::CountNodes(vtk_cell_type));
  // Initialize the VTU file:
  auto vtu = std::ofstream(filename,
      std::ios::out | (binary ? (std::ios::binary) : std::ios::out));
  vtu << "<VTKFile type=\"UnstructuredGrid\" version=\"1.0\""
      << " byte_order=" << endianness << " header_type=\"UInt64\">\n";
  vtu << "  <UnstructuredGrid>\n";
  vtu << "    <Piece NumberOfPoints=\"" << n_point
      << "\" NumberOfCells=\"" << n_cell << "\">\n";
  // Write the value of distance(x, y) as PointData:
  vtu << "      <PointData>\n";
  vtu << "        <DataArray type=\"Float64\" Name=\""
      << "DistanceToBoundary" << "\" format=" << format << ">\n";
  for (int i = 0; i < n_point; ++i) {
    vtu << distance(x[i], y[i]) << ' ';
  }
  vtu << "\n        </DataArray>\n";
  vtu << "      </PointData>\n";
  // Write point coordinates:
  vtu << "      <Points>\n";
  vtu << "        <DataArray type=\"Float64\" Name=\"Points\" "
      << "NumberOfComponents=\"3\" format=" << format << ">\n";
  for (int i = 0; i < n_point; ++i) {
    vtu << x[i] << ' ' << y[i] << ' ' << z[i] << ' ';
  }
  vtu << "\n        </DataArray>\n";
  vtu << "      </Points>\n";
  vtu << "      <Cells>\n";
  // Write cell connectivities:
  vtu << "        <DataArray type=\"Int32\" Name=\"connectivity\" "
      << "format=" << format << ">\n";
  for (auto &cell : cells) {
    for (int i = 0; i < kNodes; ++i) {
      vtu << cell[i] << ' ';
    }
  }
  vtu << "\n        </DataArray>\n";
  // Write cell connectivity offsets:
  vtu << "        <DataArray type=\"Int32\" Name=\"offsets\" "
      << "format=" << format << ">\n";
  int offset = 0;
  for (int i = 0; i < n_cell; ++i) {
    offset += kNodes;
    vtu << offset << ' ';
  }
  vtu << "\n        </DataArray>\n";
  // Write cell types:
  vtu << "        <DataArray type=\"UInt8\" Name=\"types\" "
      << "format=" << format << ">\n";
  for (int i = 0; i < n_cell; ++i) {
    vtu << static_cast<int>(vtk_cell_type) << ' ';
  }
  vtu << "\n        </DataArray>\n";
  vtu << "      </Cells>\n";
  vtu << "    </Piece>\n";
  vtu << "  </UnstructuredGrid>\n";
  vtu << "</VTKFile>\n";
}

template <std::floating_point Real>
void WriteFrame(int i_frame, int n_point,
    Real const *x, Real const *y, Real const *z,
    std::vector<std::array<int, 3>> const &faces) {
  auto vtu_name = std::string("Frame");
  vtu_name += std::to_string(i_frame);
  vtu_name += ".vtu";
  WriteVtu<Real, 3>(vtu_name, false, n_point, x, y, z,
      faces, mini::mesh::vtk::CellType::kTriangle3, distance<Real>);
  std::cout << vtu_name << " written\n";
}

template <std::floating_point Real>
using HostDynamicVector = mini::algebra::DynamicVector<Real>;

template <std::floating_point Real>
class Memory {
 public:
  Real *x_u{nullptr}, *x_v{nullptr}, *y_u{nullptr}, *y_v{nullptr};
  Real *bar_x{nullptr}, *bar_y{nullptr}, *actual_l{nullptr}, *expect_l{nullptr};

  template <std::invocable<void **, int> M>
  void Malloc(int n_byte, M &&malloc) {
    malloc((void **)&x_u, n_byte);
    malloc((void **)&x_v, n_byte);
    malloc((void **)&y_u, n_byte);
    malloc((void **)&y_v, n_byte);
    malloc((void **)&bar_x, n_byte);
    malloc((void **)&bar_y, n_byte);
    malloc((void **)&actual_l, n_byte);
    malloc((void **)&expect_l, n_byte);
  }

  template <std::invocable<void *> F>
  void Free(F &&free) {
    free(x_u);
    free(x_v);
    free(y_u);
    free(y_v);
    free(bar_x);
    free(bar_y);
    free(actual_l);
    free(expect_l);
  }
};

template <std::floating_point Real>
__device__ __host__ void GetLength(int i,
    Real const *memory_x_u, Real const *memory_x_v,
    Real const *memory_y_u, Real const *memory_y_v,
    Real *bar_x, Real *bar_y, Real *actual_l, Real *expect_l) {
  Real x_u = memory_x_u[i];
  Real x_v = memory_x_v[i];
  Real y_u = memory_y_u[i];
  Real y_v = memory_y_v[i];
  actual_l[i] = std::hypot(bar_x[i] = x_v - x_u,
                           bar_y[i] = y_v - y_u);
  expect_l[i] = scaling((x_u + x_v) / 2,
                        (y_u + y_v) / 2);
  assert(actual_l[i] >= 0);
  assert(expect_l[i] >= 0);
}

template <std::floating_point Real>
void HostGetLengths(int n_edge, Memory<Real> const &host_memory_pinned,
    Real *bar_x, Real *bar_y, Real *actual_l, Real *expect_l) {
  for (int i = 0; i < n_edge; i++) {
    GetLength(i,
        host_memory_pinned.x_u, host_memory_pinned.x_v,
        host_memory_pinned.y_u, host_memory_pinned.y_v,
        bar_x, bar_y, actual_l, expect_l);
  }
}


template <std::floating_point Real>
__global__ void DeviceGetLength(int n_edge,
    Real const *device_memory_x_u, Real const *device_memory_x_v,
    Real const *device_memory_y_u, Real const *device_memory_y_v,
    Real *bar_x, Real *bar_y, Real *actual_l, Real *expect_l) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n_edge) {
    return;
  }
  GetLength(i,
      device_memory_x_u, device_memory_x_v,
      device_memory_y_u, device_memory_y_v,
      bar_x, bar_y, actual_l, expect_l);
}

template <std::floating_point Real>
void DeviceGetLengths(int n_edge, Memory<Real> const &host_memory_pinned,
    Memory<Real> *device_memory,
    Real *bar_x, Real *bar_y, Real *actual_l, Real *expect_l) {
  // Initialize streams for concurrency:
  auto streams = new hipStream_t[NSTREAMS];
  for (int i = 0; i < NSTREAMS; i++) {
    hipStreamCreate(&streams[i]);
  } 
  int n_edge_per_stream = n_edge / NSTREAMS;
  int n_byte = n_edge_per_stream * sizeof(Real);
  // Dispatch comm and comp for n_edge_per_stream edges to NSTREAMS streams:
  for (int i_stream = 0; i_stream < NSTREAMS; i_stream++) {
    int offset = i_stream * n_edge_per_stream;
    if (i_stream + 1 == NSTREAMS) {
      n_edge_per_stream = n_edge - i_stream * n_edge_per_stream;
      n_byte = n_edge_per_stream * sizeof(Real);
    }
    // std::cout << "i_stream = " << i_stream << "\n";
    // std::cout << "  n_edge_per_stream = " << n_edge_per_stream << ", ";
    // std::cout << "  n_byte = " << n_byte << "\n";
    dim3 block = std::min(256, n_edge_per_stream);
    dim3 grid = (n_edge_per_stream + block.x - 1) / block.x;
    // std::cout << "  block.x = " << block.x << ", ";
    // std::cout << "  grid.x = " << grid.x << "\n";
    auto stream_i = streams[i_stream];
    // copy input data from host to device
    hipMemcpyAsync(device_memory->x_u + offset, host_memory_pinned.x_u + offset,
        n_byte, hipMemcpyHostToDevice, stream_i);
    hipMemcpyAsync(device_memory->x_v + offset, host_memory_pinned.x_v + offset,
        n_byte, hipMemcpyHostToDevice, stream_i);
    hipMemcpyAsync(device_memory->y_u + offset, host_memory_pinned.y_u + offset,
        n_byte, hipMemcpyHostToDevice, stream_i);
    hipMemcpyAsync(device_memory->y_v + offset, host_memory_pinned.y_v + offset,
        n_byte, hipMemcpyHostToDevice, stream_i);
    // execute the kernel
    DeviceGetLength<<< grid, block, 0, stream_i >>>(n_edge_per_stream,
        device_memory->x_u + offset, device_memory->x_v + offset,
        device_memory->y_u + offset, device_memory->y_v + offset,
        device_memory->bar_x + offset, device_memory->bar_y + offset,
        device_memory->actual_l + offset, device_memory->expect_l + offset);
    // copy output data from device to host
    hipMemcpyAsync(bar_x + offset, device_memory->bar_x + offset,
        n_byte, hipMemcpyDeviceToHost, stream_i);
    hipMemcpyAsync(bar_y + offset, device_memory->bar_y + offset,
        n_byte, hipMemcpyDeviceToHost, stream_i);
    hipMemcpyAsync(actual_l + offset, device_memory->actual_l + offset,
        n_byte, hipMemcpyDeviceToHost, stream_i);
    hipMemcpyAsync(expect_l + offset, device_memory->expect_l + offset,
        n_byte, hipMemcpyDeviceToHost, stream_i);
  }
  hipDeviceSynchronize();
  for (int i = 0; i < NSTREAMS; i++) {
    hipStreamDestroy(streams[i]);
  }
  delete[] streams;
}

template <typename T>
T Get(YAML::Node const &config, char const *key) {
  try {
    return config[key].as<T>();
  } catch (...) {
    std::fprintf(stderr, "Failed to parse the key \"%s\"\n", key);
    throw;
  }
}

int main(int argc, char *argv[]) {
  std::srand(31415926);

  using Real = double;
  using Column = HostDynamicVector<Real>;

  if (argc < 2) {
    std::cout << "usage:\n    ./distance <config.yaml>\n";
    return -1;
  }
  YAML::Node config = YAML::LoadFile(argv[1]);

  int n_point = Get<int>(config, "n_point");
  int n_frame = Get<int>(config, "n_frame");  // maximum writing step
  int n_step_per_frame = Get<int>(config, "n_step_per_frame");
  int n_step = n_step_per_frame * n_frame;  // maximum iteration step

  Real const h_0 = Get<Real>(config, "h_0");
  Real const eps = 1e-16;
  auto g_eps = h_0 * 0.001;  // for rejecting out-of-domain faces
  auto d_eps = h_0 * std::sqrt(eps);  // for finite-differencing d(x, y)

  float elapsed_time = 0.0, host_cost = 0.0, device_cost = 0.0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop); 

  hipEventRecord(start);  // timing the initialization
  // Build random points in \f$ [-1, 1]^2 \times 0 \f$
  Column x(n_point), y(n_point), z(n_point);
  x.setRandom(); y.setRandom(); z.setZero();
  // Fix corner points:
  x[0] = x[2] = x_min;
  x[1] = x[3] = x_max;
  y[0] = y[1] = y_min;
  y[2] = y[3] = y_max;

  // Reject out-of-domain (d > 0) points:
  n_point = RejectPoints(&x, &y, distance<Real>);
  assert(n_point == x.size());
  assert(n_point == y.size());

  // Pre-allocate memory:
  int n_edge_max = n_point * 4;
  auto host_memory = Memory<Real>();
  host_memory.Malloc(n_edge_max * sizeof(Real),
      [](void **ptr, int n_byte) { *ptr = std::malloc(n_byte); });
  Real *host_bar_x = host_memory.bar_x;
  Real *host_bar_y = host_memory.bar_y;
  Real *host_actual_l = host_memory.actual_l;
  Real *host_expect_l = host_memory.expect_l;

  auto host_memory_pinned = Memory<Real>();
  host_memory_pinned.Malloc(n_edge_max * sizeof(Real),
      [](void **ptr, int n_byte) { hipHostAlloc(ptr, n_byte,
          hipHostMallocDefault); });
  Real *bar_x = host_memory_pinned.bar_x;
  Real *bar_y = host_memory_pinned.bar_y;
  Real *actual_l = host_memory_pinned.actual_l;
  Real *expect_l = host_memory_pinned.expect_l;

  auto device_memory = Memory<Real>();
  device_memory.Malloc(n_edge_max * sizeof(Real),
      [](void **ptr, int n_byte) { hipMalloc(ptr, n_byte); });

  // Triangulate the points.
  using Kernel = CGAL::Simple_cartesian<Real>;
  using Point = Kernel::Point_3;
  using GeoTraits = CGAL::Projection_traits_xy_3<Kernel>;
  using Delaunay = CGAL::Delaunay_triangulation_2<GeoTraits>;

  Real const max_shift_tol = 1.e-3;  // terminate if max_shift / h_0 < this value
  Real max_shift_square = 1.e100;

  std::vector<std::array<int, 3>> faces; int n_face;
  std::vector<std::array<int, 2>> edges; int n_edge;

  auto Triangulate = [&faces, &edges, &x, &y, &z, &n_face, &n_edge, g_eps](int n_point) {
    std::cout << "Re-triangulate.\n";
    auto delaunay = Delaunay();
    for (int i = 0; i < n_point; i++) {
      delaunay.insert(Point(x[i], y[i], z[i]));
    }
    faces = GetFaces(delaunay);
    n_face = RejectFaces(&faces, x, y, distance<Real>, g_eps);
    assert(n_face <= delaunay.number_of_faces());
    edges = GetEdges(faces);
    n_edge = edges.size();
    assert(n_edge <= n_point * 4);
  };

  Real delaunay_tol = 1.e-1;  // re-triangluate if it is < max_shift
  Column x_old = x;
  Column y_old = y;
  // Eigen does not support `Vector += double` ?
  // x_old += delaunay_tol;
  // y_old += delaunay_tol;
  for (int i = 0; i < n_point; ++i) {
    x_old[i] += delaunay_tol * 2;
  }

  auto HasLargeShift = [](Real delaunay_tol,
      Column const &x_old, Column const &x,
      Column const &y_old, Column const &y) {
    Column norms = (
        (x_old - x).array().square() +
        (y_old - y).array().square()).sqrt();
    return norms.maxCoeff() > delaunay_tol;
  };

  int too_close_freq = 25;
  auto too_close = std::unordered_set<int>();

  auto RemoveTooClosePoints = [&too_close, &x, &y](int n)
      -> int {
    if (too_close.empty()) {
      return n;
    }
    int n_new = n - too_close.size();
    Column x_new(n_new), y_new(n_new);
    int i_new = 0;
    for (int i = 0; i < n; i++) {
      if (too_close.find(i) == too_close.end()) {
        x_new[i_new] = x[i];
        y_new[i_new] = y[i];
        i_new++;
      }
    }
    x = x_new;
    y = y_new;
    return n_new;
  };

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf("Initialization costs %.2f ms\n", elapsed_time);

  // The main loop:
  for (int i_step = 0; i_step <= n_step; i_step++) {
    n_point = RemoveTooClosePoints(n_point);
    if (too_close.size() || HasLargeShift(delaunay_tol, x_old, x, y_old, y)
        || i_step % n_step_per_frame == 0) {
      too_close.clear();
      x_old = x;
      y_old = y;
      Triangulate(n_point);
    }

    // Write the points and triangles.
    if (i_step % n_step_per_frame == 0) {
      int i_frame = i_step / n_step_per_frame;
      WriteFrame(i_frame, n_point, x.data(), y.data(), z.data(), faces);
    }

    // Collect point data:
    for (int i = 0; i < n_edge; i++) {
      auto [u, v] = edges[i];
      host_memory_pinned.x_u[i] = x[u];
      host_memory_pinned.x_v[i] = x[v];
      host_memory_pinned.y_u[i] = y[u];
      host_memory_pinned.y_v[i] = y[v];
    }

    // Build actual and expect lengths:
    hipEventRecord(start);
    HostGetLengths(n_edge, host_memory_pinned,
        host_bar_x, host_bar_y, host_actual_l, host_expect_l);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    host_cost += elapsed_time;

    hipEventRecord(start);
    DeviceGetLengths(n_edge, host_memory_pinned, &device_memory,
        bar_x, bar_y, actual_l, expect_l);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    device_cost += elapsed_time;

    for (int i = 0; i < n_edge; i++) {
      host_bar_x[i] = std::abs(host_bar_x[i] - bar_x[i]);
      host_bar_y[i] = std::abs(host_bar_y[i] - bar_y[i]);
      host_actual_l[i] = std::abs(host_actual_l[i] - actual_l[i]);
      host_expect_l[i] = std::abs(host_expect_l[i] - expect_l[i]);
    }

    std::cout << "|diff_bar_x| + |diff_bar_y| = " <<
        std::accumulate(host_bar_x, host_bar_x + n_edge, 0.) +
        std::accumulate(host_bar_y, host_bar_y + n_edge, 0.) << "\n";
    std::cout << "|diff_actual_l| + |diff_expect_l| = " <<
        std::accumulate(host_actual_l, host_actual_l + n_edge, 0.) +
        std::accumulate(host_expect_l, host_expect_l + n_edge, 0.) << "\n";

    auto actual_norm = std::inner_product(actual_l, actual_l + n_edge, actual_l, 0.);
    auto expect_norm = std::inner_product(expect_l, expect_l + n_edge, expect_l, 0.);
    Real norm_ratio = 1.2 * std::sqrt(actual_norm / expect_norm);
    std::cout << actual_norm << " " << expect_norm << " " << norm_ratio << "\n";
    std::for_each_n(expect_l, n_edge,
        [norm_ratio](Real &x) { x *= norm_ratio; });

    // Label too-close points:
    if ((i_step + 1) % too_close_freq == 0) {
      assert(too_close.empty());
      for (int i = 0; i < n_edge; ++i) {
        if (expect_l[i] > 2 * actual_l[i]) {
          auto [u, v] = edges[i];
          if (u >= n_fixed) {
            too_close.emplace(u);
          }
          if (v >= n_fixed) {
            too_close.emplace(v);
          }
        }
      }
      std::cout << too_close.size() << " points should be removed.\n";
      continue;
    }

    // Get forces at nodes:
    Column force_x(n_point), force_y(n_point);
    force_x.setZero(); force_y.setZero();
    for (int i = 0; i < n_edge; i++) {
      // repulsive force for compressed bars
      Real force = /* 1.0 * */std::max(0., expect_l[i] / actual_l[i] - 1);
      auto [u, v] = edges[i];
      force_x[u] -= force * bar_x[i];
      force_x[v] += force * bar_x[i];
      force_y[u] -= force * bar_y[i];
      force_y[v] += force * bar_y[i];
    }
    // No force at fixed points:
    for (int i = 0; i < n_fixed; i++) {
      force_x[i] = force_y[i] = 0.;
    }

    // Move points:
    Real delta_t = 0.2;
    Column shift_x(n_point), shift_y(n_point);
    for (int i = n_fixed; i < n_point; i++) {
      x[i] += (shift_x[i] = delta_t * force_x[i]);
      y[i] += (shift_y[i] = delta_t * force_y[i]);
    }

    // Project back out-of-domain points:
    auto out = std::vector<int>();
    for (int i = n_fixed; i < n_point; i++) {
      Real d = distance(x[i], y[i]);
      if (d <= 0) {
        continue;
      }
      // Numerical gradient of d(x, y):
      Real grad_x = (distance(x[i] + d_eps, y[i]) - d) / d_eps;
      Real grad_y = (distance(x[i], y[i] + d_eps) - d) / d_eps;
      auto fix_x = grad_x * d;
      auto fix_y = grad_y * d;
      x[i] -= fix_x;
      y[i] -= fix_y;
      shift_x[i] -= fix_x;
      shift_y[i] -= fix_y;
    }

    // Update the maximum shift:
    max_shift_square = 0.;
    for (int i = n_fixed; i < n_point; i++) {
      if (distance(x[i], y[i]) > -g_eps) {
        continue;
      }
      max_shift_square = std::max(max_shift_square,
          shift_x[i] * shift_x[i] + shift_y[i] * shift_y[i]);
    }
    auto max_shift = std::sqrt(max_shift_square) / h_0;

    std::cout << "Step " << i_step << ", n_edge = " << n_edge << ", max_shift = " << max_shift << "\n";

    if (max_shift < max_shift_tol) {
      int i_frame = (i_step + n_step_per_frame - 1) / n_step_per_frame;
      WriteFrame(i_frame, n_point, x.data(), y.data(), z.data(), faces);
      std::cout << "Converged at Step " << i_step << ", Frame " << i_frame << ".\n";
      break;
    }
  }  // main loop

  host_memory.Free([](void *ptr) { std::free(ptr); });
  host_memory_pinned.Free([](void *ptr) { hipHostFree(ptr); });
  device_memory.Free([](void *ptr) { hipFree(ptr); });

  printf("The host costs %.2f ms\n", host_cost);
  printf("The device costs %.2f ms\n", device_cost);

  return 0;
}
